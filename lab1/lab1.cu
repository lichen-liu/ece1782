#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Comment out this line to enable debug mode
// #define NDEBUG

/* time stamp function in seconds */
__host__ double getTimeStamp()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_usec / 1000000 + tv.tv_sec;
}

__host__ void initX(float *X, int numRows, int numCols)
{
    for (int i = 0; i < numRows; i++)
    {
        int ibase = i * numCols;
        for (int j = 0; j < numCols; j++)
        {
            // h_X[i,j] = (float) (i+j)/2.0;
            X[ibase + j] = (float)(i + j) / 2.0;
        }
    }
}

__host__ void initY(float *Y, int numRows, int numCols)
{
    for (int i = 0; i < numRows; i++)
    {
        int ibase = i * numCols;
        for (int j = 0; j < numCols; j++)
        {
            // h_Y[i,j] = (float) 3.25*(i+j);
            Y[ibase + j] = (float)3.25 * (i + j);
        }
    }
}

__host__ float f_siggen_reference_get(float *M, int i, int j, int numRows, int numCols)
{
    if (i < 0 || i >= numRows || j < 0 || j >= numCols)
    {
        return 0;
    }
    return M[i * numCols + j];
}

__host__ void f_siggen_reference(float *X, float *Y, float *Z, int numRows, int numCols)
{
    for (int i = 0; i < numRows; i++)
    {
        int ibase = i * numCols;
        for (int j = 0; j < numCols; j++)
        {
            // Z[i,j] = X[i-1,j] + X[i,j] + X[i+1,j] – Y[i,j-2] – Y[i,j-1] – Y[i,j]
            Z[ibase + j] =
                f_siggen_reference_get(X, i - 1, j, numRows, numCols) +
                f_siggen_reference_get(X, i, j, numRows, numCols) +
                f_siggen_reference_get(X, i + 1, j, numRows, numCols) -
                f_siggen_reference_get(Y, i, j - 2, numRows, numCols) -
                f_siggen_reference_get(Y, i, j - 1, numRows, numCols) -
                f_siggen_reference_get(Y, i, j, numRows, numCols);
        }
    }
}

__host__ int checkZ(float *E, float *A, int numRows, int numCols)
{
    for (int i = 0; i < numRows; i++)
    {
        int ibase = i * numCols;
        for (int j = 0; j < numCols; j++)
        {
            if (E[ibase + j] != A[ibase + j])
            {
                return 0;
            }
        }
    }
    return 1;
}

__global__ void f_siggen(float *X, float *Y, float *Z, int numRows, int numCols)
{
    // WIP
}

int main(int argc, char *argv[])
{
    /* Get Dimension */
    if (argc != 3)
    {
        printf("Error: The number of arguments is not exactly 2\n");
        return 0;
    }
    int numRows = atoi(argv[1]);
    int numCols = atoi(argv[2]);
    int numElem = numRows * numCols;
    int numBytes = numElem * sizeof(float);

#ifndef NDEBUG
    printf("numRows=%d, numCols=%d, numElem=%d, numBytes=%d", numRows, numCols, numElem, numBytes);
#endif

    /* Allocate Host Memory */
    float *h_X = (float *)malloc(numBytes);
    float *h_Y = (float *)malloc(numBytes);
    float *h_hZ = (float *)malloc(numBytes);
    float *h_dZ = (float *)malloc(numBytes);
    // TODO:
    // float *h_X = NULL;
    // float *h_Y = NULL;
    // float *h_hZ = (float *)malloc(numBytes);
    // float *h_dZ = NULL;
    // hipHostAlloc((void **)&h_X, numBytes, 0);
    // hipHostAlloc((void **)&h_Y, numBytes, 0);
    // hipHostAlloc((void **)&h_dZ, numBytes, hipHostMallocWriteCombined);

    /* Initialize Host Memory */
    initX(h_X, numRows, numCols);
    initY(h_Y, numRows, numCols);
    f_siggen_reference(h_X, h_Y, h_hZ, numRows, numCols);

    /* Allocate Device Memory */
    float *d_X = NULL;
    float *d_Y = NULL;
    float *d_Z = NULL;
    hipMalloc((void **)&d_X, numBytes);
    hipMalloc((void **)&d_Y, numBytes);
    hipMalloc((void **)&d_Z, numBytes);

    /* Copy Host Memory to Device Memory */
    double timestampPreCpuGpuTransfer = getTimeStamp();
    hipMemcpy(d_X, h_X, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, h_Y, numBytes, hipMemcpyHostToDevice);

    /* Run Kernel */
    double timestampPreKernel = getTimeStamp();
    dim3 gridDim;
    dim3 blockDim;
    size_t d_sizeSmem = 0;
    f_siggen<<<gridDim, blockDim, d_sizeSmem>>>(d_X, d_Y, d_Z, numRows, numCols); // WIP
    hipDeviceSynchronize();

    /* Copy Device Memory to Host Memory */
    double timestampPreGpuCpuTransfer = getTimeStamp();
    cudaMemCpy(h_dZ, d_Z, numBytes, hipMemcpyDeviceToHost);
    double timestampPostGpuCpuTransfer = getTimeStamp();

    /* Free Device Memory */
    hipFree(d_Z);
    d_Z = NULL;
    hipFree(d_Y);
    d_Y = NULL;
    hipFree(d_X);
    d_X = NULL;

    /* Clean Up Device Resource */
    hipDeviceReset();

    /* Verify Device Result with Host Result */
    int isMatching = checkZ(h_hZ, h_dZ, numRows, numCols);

    /* Output */
    if (isMatching)
    {
#ifndef NDEBUG
        printf("<total_GPU_time> <CPU_GPU_transfer_time> <kernel_time> <GPU_CPU_transfer_time> <Z-value> <nl>\n");
#endif
        float totalGpuElapased = timestampPostGpuCpuTransfer - timestampPreCpuGpuTransfer;
        float cpuGpuTransferElapsed = timestampPreKernel - timestampPreCpuGpuTransfer;
        float kernelElapsed = timestampPreGpuCpuTransfer - timestampPreKernel;
        float gpuCpuTransferElapsed = timestampPostGpuCpuTransfer - timestampPreGpuCpuTransfer;
        int zValueI = 5;
        int zValueJ = 5;
        float zValue = h_dZ[zValueI * numCols + zValueJ];
        printf("%.6f %.6f %.6f %.6f %.6f\n", totalGpuElapased, cpuGpuTransferElapsed, kernelElapsed, gpuCpuTransferElapsed, zValue);
    }
    else
    {
        printf("Error: GPU result does not with CPU result\n");
    }

    /* Free Host Memory */
    free(h_dZ);
    h_dZ = NULL;
    free(h_hZ);
    h_hZ = NULL;
    free(h_Y);
    h_Y = NULL;
    free(h_X);
    h_X = NULL;
    // TODO:
    // hipHostFree(h_dZ);
    // h_dZ = NULL;
    // free(h_hZ);
    // h_hZ = NULL;
    // hipHostFree(h_Y);
    // h_Y = NULL;
    // hipHostFree(h_X);
    // h_X = NULL;
}
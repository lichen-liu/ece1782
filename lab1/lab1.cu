
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Comment out this line to enable debug mode
// #define NDEBUG

/* time stamp function in seconds */
__host__ double getTimeStamp() {
    struct timeval  tv ; gettimeofday( &tv, NULL ) ;
    return (double) tv.tv_usec/1000000 + tv.tv_sec ;
}

__host__ void initX(float* X, int n_rows, int n_cols) {
    for(int i=0; i<n_rows; i++) {
        int ibase = i * n_cols;
        for(int j=0; j<n_cols; j++) {
            X[ibase + j] = (float) (i+j)/2.0;
        }
    }
}

__global__ void f_siggen() {
    // wip
}

int main(int argc, char *argv[]) {
    /* Get Dimension */
    if(argc != 3) {
        printf("Error: The number of arguments is not exactly 2\n");
        return 0;
    }
    int n_rows = atoi(argv[1]);
    int n_cols = atoi(argv[2]);
    int n_elem = n_rows * n_cols;

#ifndef NDEBUG
    printf("n_rows=%d, n_cols=%d, n_elem=%d", n_rows, n_cols, n_elem);
#endif

    /* Allocate and Initialize Host Memory */
    float* h_X = (float*) malloc(sizeof(float) * n_elem);
    float* h_Y = (float*) malloc(sizeof(float) * n_elem);
    float* h_hZ = (float*) malloc(sizeof(float) * n_elem);
    float* h_dZ = (float*) malloc(sizeof(float) * n_elem);

    /* Allocate Device Memory */

    /* Copy Host Memory to Device Memory */

    /* Launch Kernel */
    dim3 gridDim;
    dim3 blockDim;
    size_t d_smemSize = 0;
    f_siggen <<<gridDim, blockDim, d_smemSize>>>();

    /* Copy Device Memory to Host Memory */

    /* Clean Up Device Resource */

    /* Verify Device Result with Host Result */
    int isMatching = 1;

    /* Output */
    if(isMatching) {
#ifndef NDEBUG
        printf("<total_GPU_time> <CPU_GPU_transfer_time> <kernel_time> <GPU_CPU_transfer_time> <Z-value> <nl>\n");
#endif
        float totalGpuElapased = 0;
        float cpuGpuTransferElapsed = 0;
        float kernelElapsed = 0;
        float gpuCpuTransferElapsed = 0;
        float zValue = 0;
        printf("%.6f %.6f %.6f %.6f %.6f\n", totalGpuElapased, cpuGpuTransferElapsed, kernelElapsed, gpuCpuTransferElapsed, zValue);
    } else {
        printf("Error: GPU result does not with CPU result\n");
    }

    /* Clean Up Host Resource */
}
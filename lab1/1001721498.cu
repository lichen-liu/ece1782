
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Comment out this line to enable debug mode
// #define NDEBUG

/* time stamp function in seconds */
__host__ double getTimeStamp()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_usec / 1000000 + tv.tv_sec;
}

__host__ void initX(float *X, int numRows, int numCols)
{
    int lastIBase = (numRows + 1) * numCols;
    for (int j = 0; j < numCols; j++)
    {
        X[j] = 0;
        X[lastIBase + j] = 0;
    }
    for (int i = 0; i < numRows; i++)
    {
        int iBase = (i + 1) * numCols;
        for (int j = 0; j < numCols; j++)
        {
            // h_X_old[i,j] = (float) (i+j)/2.0;
            X[iBase + j] = (float)(i + j) / 2.0;
        }
    }
}

__host__ void initY(float *Y, int numRows, int numCols)
{
    for (int i = 0; i < numRows; i++)
    {
        int iBase = i * (numCols + 2);
        Y[iBase] = 0;
        Y[iBase + 1] = 0;
        for (int j = 0; j < numCols; j++)
        {
            // h_Y_old[i,j] = (float) 3.25*(i+j);
            Y[iBase + j + 2] = (float)3.25 * (i + j);
        }
    }
}

#define H_ADJ_INDEX_X(i, j) ((i) + 1) * numCols + (j)
#define H_ADJ_INDEX_Y(i, j) (i) * (numCols + 2) + (j) + 2
#define H_INDEX(i, j) (i) * numCols + (j)

__host__ void f_siggen_reference(float *X, float *Y, float *Z, int numRows, int numCols)
{
    for (int i = 0; i < numRows; i++)
    {
        for (int j = 0; j < numCols; j++)
        {
            // Z[i,j] = X[i-1,j] + X[i,j] + X[i+1,j] – Y[i,j-2] – Y[i,j-1] – Y[i,j]
            Z[H_INDEX(i, j)] =
                X[H_ADJ_INDEX_X(i - 1, j)] +
                X[H_ADJ_INDEX_X(i, j)] +
                X[H_ADJ_INDEX_X(i + 1, j)] -
                Y[H_ADJ_INDEX_Y(i, j - 2)] -
                Y[H_ADJ_INDEX_Y(i, j - 1)] -
                Y[H_ADJ_INDEX_Y(i, j)];
        }
    }
}

__host__ int checkZ(float *E, float *A, int numRows, int numCols)
{
    for (int i = 0; i < numRows; i++)
    {
        int ibase = i * numCols;
        for (int j = 0; j < numCols; j++)
        {
            if (E[ibase + j] != A[ibase + j])
            {
                printf("(i=%d, j=%d) Expected=%f Actual=%f\n", i, j, E[ibase + j], A[ibase + j]);
                return 0;
            }
        }
    }
    return 1;
}

__global__ void f_siggen(float *X, float *Y, float *Z, int numRows, int numCols, int smemNumElemX)
{
    extern __shared__ float s_data[];
    float *s_XT = s_data; // blockDim.x * (blockDim.y + 2);
    int s_XTWidth = (blockDim.y + 2);
    // int s_XTHeight = blockDim.x;
    float *s_Y = s_XT + smemNumElemX; // (blockDim.x + 2) * blockDim.y;

    /* Global Coordinate */
    int globalX = blockDim.x * blockIdx.x + threadIdx.x;
    int globalY = blockDim.y * blockIdx.y + threadIdx.y;
    int globalIdx = globalY * numCols + globalX;
    int globalXIdx = (globalY + 1) * numCols + globalX;
    int globalYIdx = globalY * (numCols + 2) + globalX + 2;

    if (globalX >= numCols || globalY >= numRows)
        return;

    /* Set Up s_XT */
    int s_XTx = threadIdx.y + 1;
    int s_XTy = threadIdx.x;
    int s_XTIdx = s_XTy * s_XTWidth + s_XTx;
    if (threadIdx.y == 0)
    {
        s_XT[s_XTIdx - 1] = X[globalXIdx - numCols];
    }
    if (threadIdx.y == blockDim.y - 1 || globalY == numRows - 1)
    {
        s_XT[s_XTIdx + 1] = X[globalXIdx + numCols];
    }
    s_XT[s_XTIdx] = X[globalXIdx];

    /* Set Up s_Y */
    int s_Yx = threadIdx.x + 2;
    int s_Yy = threadIdx.y;
    int s_YIdx = s_Yy * (blockDim.x + 2) + s_Yx;
    if (threadIdx.x == 0)
    {
        s_Y[s_YIdx - 2] = Y[globalYIdx - 2];
        s_Y[s_YIdx - 1] = Y[globalYIdx - 1];
    }
    s_Y[s_YIdx] = Y[globalYIdx];

    /* Wait for All to Set Up s_XT and s_Y */
    __syncthreads();

    /* Write Output */
    Z[globalIdx] = s_XT[s_XTIdx - 1] + s_XT[s_XTIdx] + s_XT[s_XTIdx + 1] - s_Y[s_YIdx - 2] - s_Y[s_YIdx - 1] - s_Y[s_YIdx];
}

int main(int argc, char *argv[])
{
    int error = 0;
    /* Get Dimension */
    if (argc != 3)
    {
        printf("Error: The number of arguments is not exactly 2\n");
        return 0;
    }
    int numRows = atoi(argv[1]);
    int numCols = atoi(argv[2]);
    size_t numElem = numRows * numCols;
    size_t numBytes = numElem * sizeof(float);

    int numRowsX = numRows + 2;
    int numColsX = numCols;
    size_t numElemX = numRowsX * numColsX;
    size_t numBytesX = numElemX * sizeof(float);

    int numRowsY = numRows;
    int numColsY = numCols + 2;
    size_t numElemY = numRowsY * numColsY;
    size_t numBytesY = numElemY * sizeof(float);

#ifndef NDEBUG
    printf("numRows=%d, numCols=%d, numElem=%ld, numBytes=%ld\n", numRows, numCols, numElem, numBytes);
    printf("numRowsX=%d, numColsX=%d, numElemX=%ld, numBytesX=%ld\n", numRowsX, numColsX, numElemX, numBytesX);
    printf("numRowsY=%d, numColsY=%d, numElemY=%ld, numBytesY=%ld\n", numRowsY, numColsY, numElemY, numBytesY);
#endif

    /* Allocate Host Memory */
    float *h_X = NULL;
    float *h_Y = NULL;
    float *h_hZ = (float *)malloc(numBytes);
    float *h_dZ = NULL;
    error = error || hipHostAlloc((void **)&h_X, numBytesX, 0);
    error = error || hipHostAlloc((void **)&h_Y, numBytesY, 0);
    error = error || hipHostAlloc((void **)&h_dZ, numBytes, 0);
    if (error)
    {
        printf("Error: hipHostAlloc returns error\n");
        return 0;
    }

    /* Initialize Host Memory */
    initX(h_X, numRows, numCols);
    initY(h_Y, numRows, numCols);
#ifndef NDEBUG
    double timestampPreCpuKernel = getTimeStamp();
#endif
    f_siggen_reference(h_X, h_Y, h_hZ, numRows, numCols);
#ifndef NDEBUG
    double timestampPostCpuKernel = getTimeStamp();
    printf("CPU=%.6fsec\n", timestampPostCpuKernel - timestampPreCpuKernel);
#endif

    /* Allocate Device Memory */
    float *d_X = NULL;
    float *d_Y = NULL;
    float *d_Z = NULL;
    error = error || hipMalloc((void **)&d_X, numBytes);
    error = error || hipMalloc((void **)&d_Y, numBytes);
    error = error || hipMalloc((void **)&d_Z, numBytes);
    if (error)
    {
        printf("Error: hipMalloc returns error\n");
        return 0;
    }

    /* Copy Host Memory to Device Memory */
    double timestampPreCpuGpuTransfer = getTimeStamp();
    error = error || hipMemcpy(d_X, h_X, numBytesX, hipMemcpyHostToDevice);
    error = error || hipMemcpy(d_Y, h_Y, numBytesY, hipMemcpyHostToDevice);
    if (error)
    {
        printf("Error: hipMemcpy returns error\n");
        return 0;
    }

    /* Run Kernel */
    double timestampPreKernel = getTimeStamp();
    dim3 d_blockDim;
    d_blockDim.x = 32;
    d_blockDim.y = 32;
    dim3 d_gridDim;
    d_gridDim.x = (numCols - 1) / d_blockDim.x + 1;
    d_gridDim.y = (numRows - 1) / d_blockDim.y + 1;
    int d_smemNumElemX = d_blockDim.x * (d_blockDim.y + 2);
    int d_smemNumElemY = (d_blockDim.x + 2) * d_blockDim.y;
    size_t d_smemNumBytes = (d_smemNumElemX + d_smemNumElemY) * sizeof(float);
    f_siggen<<<d_gridDim, d_blockDim, d_smemNumBytes>>>(d_X, d_Y, d_Z, numRows, numCols, d_smemNumElemX);
    hipDeviceSynchronize();

    /* Copy Device Memory to Host Memory */
    double timestampPreGpuCpuTransfer = getTimeStamp();
    error = error || hipMemcpy(h_dZ, d_Z, numBytes, hipMemcpyDeviceToHost);
    if (error)
    {
        printf("Error: hipMemcpy returns error\n");
        return 0;
    }
    double timestampPostGpuCpuTransfer = getTimeStamp();

    /* Free Device Memory */
    hipFree(d_Z);
    d_Z = NULL;
    hipFree(d_Y);
    d_Y = NULL;
    hipFree(d_X);
    d_X = NULL;

    /* Verify Device Result with Host Result */
    error = error || !checkZ(h_hZ, h_dZ, numRows, numCols);

    /* Output */
#ifndef NDEBUG
    printf("d_gridDim=(%d, %d), d_blockDim=(%d, %d), d_smemNumBytes=%ld\n", d_gridDim.x, d_gridDim.y, d_blockDim.x, d_blockDim.y, d_smemNumBytes);
#endif

    if (!error)
    {
        // #ifndef NDEBUG
        //         printf("<total_GPU_time> <CPU_GPU_transfer_time> <kernel_time> <GPU_CPU_transfer_time> <Z-value> <nl>\n");
        // #endif
        float totalGpuElapased = timestampPostGpuCpuTransfer - timestampPreCpuGpuTransfer;
        float cpuGpuTransferElapsed = timestampPreKernel - timestampPreCpuGpuTransfer;
        float kernelElapsed = timestampPreGpuCpuTransfer - timestampPreKernel;
        float gpuCpuTransferElapsed = timestampPostGpuCpuTransfer - timestampPreGpuCpuTransfer;
        int zValueI = 5;
        int zValueJ = 5;
        float zValue = h_dZ[H_INDEX(zValueI, zValueJ)];
        printf("%.6f %.6f %.6f %.6f %.6f\n", totalGpuElapased, cpuGpuTransferElapsed, kernelElapsed, gpuCpuTransferElapsed, zValue);
    }
    else
    {
        printf("Error: GPU result does not with CPU result\n");
#ifndef NDEBUG
        for (int i = 0; i < 4; i++)
        {
            for (int j = 0; j < 4; j++)
            {
                printf("(i=%d, j=%d), CPU=%.6f, GPU=%.6f, X=%.6f, Y=%.6f\n", i, j, h_hZ[H_INDEX(i, j)], h_dZ[H_INDEX(i, j)], h_X[H_ADJ_INDEX_X(i, j)], h_Y[H_ADJ_INDEX_Y(i, j)]);
            }
        }
#endif
    }

    /* Free Host Memory */
    hipHostFree(h_dZ);
    h_dZ = NULL;
    free(h_hZ);
    h_hZ = NULL;
    hipHostFree(h_Y);
    h_Y = NULL;
    hipHostFree(h_X);
    h_X = NULL;

    /* Clean Up Device Resource */
    hipDeviceReset();
}
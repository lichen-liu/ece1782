
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Comment out this line to enable debug mode
// #define NDEBUG

/* time stamp function in milliseconds */
__host__ long getTimeStamp()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (long)tv.tv_usec / 1000 + tv.tv_sec * 1000;
}

__host__ void initB(float *B, int nB)
{
    for (int i = 0; i < nB; i++)
    {
        int iIndex = i * nB * nB;
        for (int j = 0; j < nB; j++)
        {
            int ijIndex = iIndex + j * nB;
            for (int k = 0; k < nB; k++)
            {
                int ijkIndex = ijIndex + k;
                if (i == 0 || j == 0 || k == 0)
                {
                    B[ijkIndex] = 0;
                }
                else
                {
                    B[ijkIndex] = ((i - 1 + j - 1 + k - 1) % 10) * (float)1.1;
                }
            }
        }
    }
}

#define h_getB(B, nB, i, j, k) B[((i) + 1) * nB * nB + ((j) + 1) * nB + ((k) + 1)]

__host__ void jacobiRelaxationReference(float *A, float *B, int n)
{
    int nB = n + 1;
    for (int i = 0; i < n; i++)
    {
        int iIndex = i * n * n;
        for (int j = 0; j < n; j++)
        {
            int ijIndex = iIndex + j * n;
            for (int k = 0; k < n; k++)
            {
                int ijkIndex = ijIndex + k;
                if (i >= n - 1 || j >= n - 1 || k >= n - 1)
                {
                    A[ijkIndex] = 0.0;
                }
                else
                {
                    A[ijkIndex] = (float)0.8 * (h_getB(B, nB, i - 1, j, k) +
                                                h_getB(B, nB, i + 1, j, k) +
                                                h_getB(B, nB, i, j - 1, k) +
                                                h_getB(B, nB, i, j + 1, k) +
                                                h_getB(B, nB, i, j, k - 1) +
                                                h_getB(B, nB, i, j, k + 1));
                }
            }
        }
    }
}

__host__ int checkA(float *Expected, float *Actual, int n)
{
    for (int i = 0; i < n; i++)
    {
        int iIndex = i * n * n;
        for (int j = 0; j < n; j++)
        {
            int ijIndex = iIndex + j * n;
            for (int k = 0; k < n; k++)
            {
                int ijkIndex = ijIndex + k;
                if (Expected[ijkIndex] != Actual[ijkIndex])
                {
#ifndef NDEBUG
                    printf("(i=%d, j=%d, k=%d) Expected=%f Actual=%f\n", i, j, k, Expected[ijkIndex], Actual[ijkIndex]);
#endif
                    return 0;
                }
            }
        }
    }
    return 1;
}

__host__ float sumA(float *A, int n)
{
    float sum = 0;
    for (int i = 0; i < n; i++)
    {
        int iIndex = i * n * n;
        for (int j = 0; j < n; j++)
        {
            int ijIndex = iIndex + j * n;
            for (int k = 0; k < n; k++)
            {
                int ijkIndex = ijIndex + k;
                sum += A[ijkIndex] * (((i + j + k) % 10) ? 1 : -1);
            }
        }
    }
    return sum;
}

__global__ void jacobiRelaxation(float *A, float *B, int n)
{
    extern __shared__ float s_data[];

    /* Global Index */
    int globalK = blockDim.x * blockIdx.x + threadIdx.x;
    int globalJ = blockDim.y * blockIdx.y + threadIdx.y;
    int globalI = blockDim.z * blockIdx.z + threadIdx.z;
    int globalIdx = globalI * n * n + globalJ * n + globalK;

    int nB = n + 1;
    int sizePerGlobalBI = nB * nB;
    int sizePerGlobalBJ = nB;
    int globalBIIndex = (globalI + 1) * sizePerGlobalBI;
    int globalBIJIndex = globalBIIndex + (globalJ + 1) * sizePerGlobalBJ;
    int globalBIdx = globalBIJIndex + (globalK + 1);

    if (globalK >= n || globalJ >= n || globalI >= n)
    {
        return;
    }

    /* Local Index */
    // int sizeI = blockDim.z + 2;
    int sizeJ = blockDim.y + 2;
    int sizeK = blockDim.x + 2;
    int sizePerLocalI = sizeJ * sizeK;
    int sizePerLocalJ = sizeK;
    int localIIndex = (threadIdx.z + 1) * sizePerLocalI;
    int localIJIndex = localIIndex + (threadIdx.y + 1) * sizePerLocalJ;
    int localIdx = localIJIndex + (threadIdx.x + 1);

    s_data[localIdx] = B[globalBIdx];

    if (threadIdx.z == 0)
    {
        s_data[localIdx - sizePerLocalI] = B[globalBIdx - sizePerGlobalBI];
        s_data[localIdx + blockDim.z * sizePerLocalI] = B[globalBIdx + blockDim.z * sizePerGlobalBI];
    }
    if (threadIdx.y == 0)
    {
        s_data[localIdx - sizePerLocalJ] = B[globalBIdx - sizePerGlobalBJ];
        s_data[localIdx + blockDim.y * sizePerLocalJ] = B[globalBIdx + blockDim.y * sizePerGlobalBJ];
    }
    if (threadIdx.x == 0)
    {
        s_data[localIdx - 1] = B[globalBIdx - 1];
        s_data[localIdx + blockDim.x] = B[globalBIdx + blockDim.x];
    }

    __syncthreads();

    if (globalK == n - 1 || globalJ == n - 1 || globalI == n - 1)
    {
        A[globalIdx] = 0;
    }
    else
    {
        A[globalIdx] = (float)0.8 * (s_data[localIdx - sizePerLocalI] +
                                     s_data[localIdx + sizePerLocalI] +
                                     s_data[localIdx - sizePerLocalJ] +
                                     s_data[localIdx + sizePerLocalJ] +
                                     s_data[localIdx - 1] +
                                     s_data[localIdx + 1]);
    }
}

int main(int argc, char *argv[])
{
    int error = 0;
    /* Get Dimension */
    if (argc != 2)
    {
        printf("Error: The number of arguments is not exactly 1\n");
        return 0;
    }
    int n = atoi(argv[1]);
    size_t numElem = n * n * n;
    size_t numBytes = numElem * sizeof(float);

    int nB = n + 1;
    size_t numElemB = nB * nB * nB;
    size_t numBytesB = numElemB * sizeof(float);

#ifndef NDEBUG
    printf("n=%d, numElem=%ld, numBytes=%ld\n", n, numElem, numBytes);
    printf("nB=%d, numElemB=%ld, numBytesB=%ld\n", nB, numElemB, numBytesB);
#endif

    /* Allocate Host Memory */
    float *h_B = NULL;
    error = error || hipHostAlloc((void **)&h_B, numBytesB, 0);
    float *h_hA = (float *)malloc(numBytes);
    float *h_dA = NULL;
    error = error || hipHostAlloc((void **)&h_dA, numBytes, 0);
    if (error)
    {
        printf("Error: hipHostAlloc returns error\n");
        return 0;
    }

    /* Initialize Host Memory */
    initB(h_B, nB);
#ifndef NDEBUG
    long timestampPreCpuKernel = getTimeStamp();
#endif
    jacobiRelaxationReference(h_hA, h_B, n);
#ifndef NDEBUG
    long timestampPostCpuKernel = getTimeStamp();
    printf("CPU: %lf %ld\n", sumA(h_hA, n), timestampPostCpuKernel - timestampPreCpuKernel);
#endif

    /* Allocate Device Memory */
    float *d_B = NULL;
    error = error || hipMalloc((void **)&d_B, numBytesB);
    float *d_A = NULL;
    error = error || hipMalloc((void **)&d_A, numBytes);
    if (error)
    {
        printf("Error: hipMalloc returns error\n");
        return 0;
    }

    /* Copy Host Memory to Device Memory */
    long timestampPreCpuGpuTransfer = getTimeStamp();
    error = error || hipMemcpy(d_B, h_B, numBytesB, hipMemcpyHostToDevice);
    if (error)
    {
        printf("Error: hipMemcpy returns error\n");
        return 0;
    }

    /* Run Kernel */
    long timestampPreKernel = getTimeStamp();
    dim3 d_blockDim;
    d_blockDim.x = 32;
    d_blockDim.y = 32;
    d_blockDim.z = 1;
    dim3 d_gridDim;
    d_gridDim.x = (n - 1) / d_blockDim.x + 1;
    d_gridDim.y = (n - 1) / d_blockDim.y + 1;
    d_gridDim.z = (n - 1) / d_blockDim.z + 1;
    int d_smemNumElem = (d_blockDim.x + 2) * (d_blockDim.y + 2) * (d_blockDim.z + 2);
    size_t d_smemNumBytes = d_smemNumElem * sizeof(float);
    jacobiRelaxation<<<d_gridDim, d_blockDim, d_smemNumBytes>>>(d_A, d_B, n);
    hipDeviceSynchronize();

    /* Copy Device Memory to Host Memory */
    long timestampPreGpuCpuTransfer = getTimeStamp();
    error = error || hipMemcpy(h_dA, d_A, numBytes, hipMemcpyDeviceToHost);
    if (error)
    {
        printf("Error: hipMemcpy returns error\n");
        return 0;
    }
    long timestampPostGpuCpuTransfer = getTimeStamp();

    /* Free Device Memory */
    hipFree(d_A);
    d_A = NULL;
    hipFree(d_B);
    d_B = NULL;

    /* Verify Device Result with Host Result */
    error = error || !checkA(h_hA, h_dA, n);

/* Output */
#ifndef NDEBUG
    printf("d_gridDim=(%d, %d, %d), d_blockDim=(%d, %d, %d), d_smemNumBytes=%ld\n", d_gridDim.x, d_gridDim.y, d_gridDim.z, d_blockDim.x, d_blockDim.y, d_blockDim.z, d_smemNumBytes);
#endif

    if (!error)
    {
        float aValue = sumA(h_dA, n);
        long totalGpuElapased = timestampPostGpuCpuTransfer - timestampPreCpuGpuTransfer;
        printf("%lf %ld\n", aValue, totalGpuElapased);
#ifndef NDEBUG
        printf("<total_GPU_time> <CPU_GPU_transfer_time> <kernel_time> <GPU_CPU_transfer_time> <A-value> <nl>\n");
        long cpuGpuTransferElapsed = timestampPreKernel - timestampPreCpuGpuTransfer;
        long kernelElapsed = timestampPreGpuCpuTransfer - timestampPreKernel;
        long gpuCpuTransferElapsed = timestampPostGpuCpuTransfer - timestampPreGpuCpuTransfer;
        printf("%ld %ld %ld %ld %.6f\n", totalGpuElapased, cpuGpuTransferElapsed, kernelElapsed, gpuCpuTransferElapsed, aValue);
#endif
    }
    else
    {
        printf("Error: GPU result does not with CPU result\n");
    }

    /* Free Host Memory */
    hipHostFree(h_dA);
    h_dA = NULL;
    free(h_hA);
    h_hA = NULL;
    hipHostFree(h_B);
    h_B = NULL;

    /* Clean Up Device Resource */
    hipDeviceReset();
}
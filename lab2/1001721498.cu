
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

// Comment out this line to enable debug mode
#define NDEBUG

/* time stamp function in milliseconds */
__host__ double getTimeStamp()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_usec / 1000000 + tv.tv_sec;
}

__host__ void initB(float *B, int nB)
{
    for (int i = 0; i < nB; i++)
    {
        int iIndex = i * nB * nB;
        for (int j = 0; j < nB; j++)
        {
            int ijIndex = iIndex + j * nB;
            for (int k = 0; k < nB; k++)
            {
                int ijkIndex = ijIndex + k;
                if (i == 0 || j == 0 || k == 0)
                {
                    B[ijkIndex] = 0;
                }
                else
                {
                    B[ijkIndex] = ((i - 1 + j - 1 + k - 1) % 10) * (float)1.1;
                }
            }
        }
    }
}

#define h_getB(B, nB, i, j, k) B[((i) + 1) * nB * nB + ((j) + 1) * nB + ((k) + 1)]

__host__ void jacobiRelaxationReference(float *A, float *B, int n)
{
    int nB = n + 1;
    for (int i = 0; i < n; i++)
    {
        int iIndex = i * n * n;
        for (int j = 0; j < n; j++)
        {
            int ijIndex = iIndex + j * n;
            for (int k = 0; k < n; k++)
            {
                int ijkIndex = ijIndex + k;
                if (i >= n - 1 || j >= n - 1 || k >= n - 1)
                {
                    A[ijkIndex] = 0.0;
                }
                else
                {
                    A[ijkIndex] = (float)0.8 * (h_getB(B, nB, i - 1, j, k) +
                                                h_getB(B, nB, i + 1, j, k) +
                                                h_getB(B, nB, i, j - 1, k) +
                                                h_getB(B, nB, i, j + 1, k) +
                                                h_getB(B, nB, i, j, k - 1) +
                                                h_getB(B, nB, i, j, k + 1));
                }
            }
        }
    }
}

__host__ int checkA(float *Expected, float *Actual, int n)
{
    for (int i = 0; i < n; i++)
    {
        int iIndex = i * n * n;
        for (int j = 0; j < n; j++)
        {
            int ijIndex = iIndex + j * n;
            for (int k = 0; k < n; k++)
            {
                int ijkIndex = ijIndex + k;
                if (Expected[ijkIndex] != Actual[ijkIndex])
                {
#ifndef NDEBUG
                    printf("(i=%d, j=%d, k=%d) Expected=%f Actual=%f\n", i, j, k, Expected[ijkIndex], Actual[ijkIndex]);
#endif
                    return 0;
                }
            }
        }
    }
    return 1;
}

__host__ double sumA(float *A, int n)
{
    double sum = 0;
    for (int i = 0; i < n; i++)
    {
        int iIndex = i * n * n;
        for (int j = 0; j < n; j++)
        {
            int ijIndex = iIndex + j * n;
            for (int k = 0; k < n; k++)
            {
                int ijkIndex = ijIndex + k;
                sum += A[ijkIndex] * (((i + j + k) % 10) ? 1 : -1);
            }
        }
    }
    return sum;
}

__global__ void jacobiRelaxation(float *A, float *B, int n, int startingI)
{
    extern __shared__ float s_data[];

    /* Global Index */
    int globalK = blockDim.x * blockIdx.x + threadIdx.x;
    int globalJ = blockDim.y * blockIdx.y + threadIdx.y;
    int globalI = blockDim.z * blockIdx.z + threadIdx.z + startingI;
    int globalIdx = globalI * n * n + globalJ * n + globalK;

    int nB = n + 1;
    int sizePerGlobalBI = nB * nB;
    int sizePerGlobalBJ = nB;
    int globalBIIndex = (globalI + 1) * sizePerGlobalBI;
    int globalBIJIndex = globalBIIndex + (globalJ + 1) * sizePerGlobalBJ;
    int globalBIdx = globalBIJIndex + (globalK + 1);

    if (globalK >= n || globalJ >= n || globalI >= n)
    {
        return;
    }

    /* Local Index */
    // int sizeI = blockDim.z + 2;
    int sizeJ = blockDim.y + 2;
    int sizeK = blockDim.x + 2;
    int sizePerLocalI = sizeJ * sizeK;
    int sizePerLocalJ = sizeK;
    int localIIndex = (threadIdx.z + 1) * sizePerLocalI;
    int localIJIndex = localIIndex + (threadIdx.y + 1) * sizePerLocalJ;
    int localIdx = localIJIndex + (threadIdx.x + 1);

    s_data[localIdx] = B[globalBIdx];

    if (threadIdx.z == 0)
    {
        s_data[localIdx - sizePerLocalI] = B[globalBIdx - sizePerGlobalBI];
        s_data[localIdx + blockDim.z * sizePerLocalI] = B[globalBIdx + blockDim.z * sizePerGlobalBI];
    }
    if (threadIdx.y == 0)
    {
        s_data[localIdx - sizePerLocalJ] = B[globalBIdx - sizePerGlobalBJ];
        s_data[localIdx + blockDim.y * sizePerLocalJ] = B[globalBIdx + blockDim.y * sizePerGlobalBJ];
    }
    if (threadIdx.x == 0)
    {
        s_data[localIdx - 1] = B[globalBIdx - 1];
        s_data[localIdx + blockDim.x] = B[globalBIdx + blockDim.x];
    }

    __syncthreads();

    if (globalK == n - 1 || globalJ == n - 1 || globalI == n - 1)
    {
        A[globalIdx] = 0;
    }
    else
    {
        A[globalIdx] = (float)0.8 * (s_data[localIdx - sizePerLocalI] +
                                     s_data[localIdx + sizePerLocalI] +
                                     s_data[localIdx - sizePerLocalJ] +
                                     s_data[localIdx + sizePerLocalJ] +
                                     s_data[localIdx - 1] +
                                     s_data[localIdx + 1]);
    }
}

int main(int argc, char *argv[])
{
    int error = 0;

    /* Get Dimension */
    if (argc != 2)
    {
        printf("Error: The number of arguments is not exactly 1\n");
        return 0;
    }
    int n = atoi(argv[1]);
    size_t numElem = n * n * n;
    size_t numBytes = numElem * sizeof(float);

    int nB = n + 1;
    size_t numElemB = nB * nB * nB;
    size_t numBytesB = numElemB * sizeof(float);

#ifndef NDEBUG
    printf("n=%d, numElem=%ld, numBytes=%ld\n", n, numElem, numBytes);
    printf("nB=%d, numElemB=%ld, numBytesB=%ld\n", nB, numElemB, numBytesB);
#endif

    /* Allocate Host Memory */
    float *h_B = NULL;
    error = error || hipHostAlloc((void **)&h_B, numBytesB, 0);
#ifndef NDEBUG
    float *h_hA = (float *)malloc(numBytes);
#endif
    float *h_dA = NULL;
    error = error || hipHostAlloc((void **)&h_dA, numBytes, 0);
    if (error)
    {
        printf("Error: hipHostAlloc returns error\n");
        return 0;
    }

    /* Initialize Host Memory */
    initB(h_B, nB);
#ifndef NDEBUG
    double timestampPreCpuKernel = getTimeStamp();
    jacobiRelaxationReference(h_hA, h_B, n);
    double timestampPostCpuKernel = getTimeStamp();
    printf("CPU: %lf %ld\n", sumA(h_hA, n), (long)ceil(1000*(timestampPostCpuKernel - timestampPreCpuKernel)));
#endif

    /* Allocate Device Memory */
    float *d_B = NULL;
    error = error || hipMalloc((void **)&d_B, numBytesB);
    float *d_A = NULL;
    error = error || hipMalloc((void **)&d_A, numBytes);
    if (error)
    {
        printf("Error: hipMalloc returns error\n");
        return 0;
    }

    /* Configuration */
#define NUM_STREAM 2
    int nIStreams[NUM_STREAM];
    for (int i = 0; i < NUM_STREAM; i++)
    {
        nIStreams[i] = n / NUM_STREAM;
    }
    nIStreams[NUM_STREAM - 1] += n % NUM_STREAM;

    dim3 d_blockDim;
    d_blockDim.x = 32;
    d_blockDim.y = 32;
    d_blockDim.z = 1; // must be 1

    dim3 d_gridDimStreams[NUM_STREAM];
    for (int i = 0; i < NUM_STREAM; i++)
    {
        d_gridDimStreams[i].x = (n - 1) / d_blockDim.x + 1;
        d_gridDimStreams[i].y = (n - 1) / d_blockDim.y + 1;
        d_gridDimStreams[i].z = (nIStreams[i] - 1) / d_blockDim.z + 1;
    }

    /* Create NUM_STREAM Streams */
    hipStream_t d_streams[NUM_STREAM];
    for (int i = 0; i < NUM_STREAM; i++)
    {
        error = error || hipStreamCreate(&d_streams[i]);
    }
    if (error)
    {
        printf("Error: hipStreamCreate returns error\n");
        return 0;
    }

    // TIMER BEGIN
    /* Copy Host Memory to Device Memory */
    double timestampPreCpuGpuTransfer = getTimeStamp();

    size_t numElemBStream1 = 0;
    if (NUM_STREAM != 1)
    {
        numElemBStream1 = (nIStreams[0] + 1 + 1) * nB * nB;
    }
    else
    {
        numElemBStream1 = (nIStreams[0] + 1) * nB * nB;
    }
    error = error || hipMemcpyAsync(d_B, h_B, numElemBStream1 * sizeof(float), hipMemcpyHostToDevice, d_streams[0]);
    if (NUM_STREAM != 1)
    {
        hipStreamSynchronize(d_streams[0]);
    }

    int numElemBStreams = numElemBStream1;
    for (int i = 1; i < NUM_STREAM; i++)
    {
        int nBIStreami = nIStreams[i];
        size_t numElemBStreami = ((i == NUM_STREAM - 1) ? nBIStreami - 1 : nBIStreami) * nB * nB;
        error = error || hipMemcpyAsync(d_B + numElemBStreams, h_B + numElemBStreams, numElemBStreami * sizeof(float), hipMemcpyHostToDevice, d_streams[i]);
        numElemBStreams += numElemBStreami;
        if (i != NUM_STREAM - 1)
        {
            // Synchronize between cudaMemcpyAsync
            hipStreamSynchronize(d_streams[i]);
        }
    }

    if (numElemBStreams != numElemB)
    {
        printf("Error: hipMemcpyAsync does not cover entire B (%ld != %ld)\n", numElemBStreams, numElemB);
        return 0;
    }
    if (error)
    {
        printf("Error: hipMemcpyAsync B returns error %d\n", error);
        return 0;
    }

    /* Run Kernel */
    int d_smemNumElem = (d_blockDim.x + 2) * (d_blockDim.y + 2) * (d_blockDim.z + 2);
    size_t d_smemNumBytes = d_smemNumElem * sizeof(float);
    size_t d_startingI = 0;
    for (int i = 0; i < NUM_STREAM; i++)
    {
        jacobiRelaxation<<<d_gridDimStreams[i], d_blockDim, d_smemNumBytes, d_streams[i]>>>(d_A, d_B, n, d_startingI);
        d_startingI += nIStreams[i];
    }

    /* Copy Device Memory to Host Memory */
    size_t numElemAStreams = 0;
    for (int i = 0; i < NUM_STREAM; i++)
    {
        size_t numElemAStreami = nIStreams[i] * n * n;
        error = error || hipMemcpyAsync(h_dA + numElemAStreams, d_A + numElemAStreams, numElemAStreami * sizeof(float), hipMemcpyDeviceToHost, d_streams[i]);
        numElemAStreams += numElemAStreami;
    }

    if (numElemAStreams != numElem)
    {
        printf("Error: hipMemcpyAsync does not cover entire A\n");
        return 0;
    }
    if (error)
    {
        printf("Error: hipMemcpyAsync A returns error %d\n", error);
        return 0;
    }

    /* Synchronize Streams */
    for (int i = 0; i < NUM_STREAM; i++)
    {
        hipStreamSynchronize(d_streams[i]);
    }

    double timestampPostGpuCpuTransfer = getTimeStamp();
    // TIMER END

    /* Free Device Memory */
    hipFree(d_A);
    d_A = NULL;
    hipFree(d_B);
    d_B = NULL;

    /* Output */
    double aValue = sumA(h_dA, n);
    long totalGpuElapased = (long)ceil(1000*(timestampPostGpuCpuTransfer - timestampPreCpuGpuTransfer));
    printf("%lf %ld\n", aValue, totalGpuElapased);
    
#ifndef NDEBUG
    for (int i = 0; i < NUM_STREAM; i++)
    {
        printf("d_gridDimStream%d=(%d, %d, %d), d_blockDim=(%d, %d, %d), d_smemNumBytes=%ld\n", i, d_gridDimStreams[i].x, d_gridDimStreams[i].y, d_gridDimStreams[i].z, d_blockDim.x, d_blockDim.y, d_blockDim.z, d_smemNumBytes);
    }

    /* Verify Device Result with Host Result */
    error = error || !checkA(h_hA, h_dA, n);
    if(error)
    {
        printf("Error: GPU result does not with CPU result\n");
    }
#endif
    
    /* Free Host Memory */
    hipHostFree(h_dA);
    h_dA = NULL;
#ifndef NDEBUG
    free(h_hA);
    h_hA = NULL;
#endif
    hipHostFree(h_B);
    h_B = NULL;

    /* Clean Up Device Resource */
    hipDeviceReset();
}
